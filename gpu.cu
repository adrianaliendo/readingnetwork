/* 
 * File:   gpu.cpp
 * Author: aliendo
 * 
 * Created on 26 de diciembre de 2013, 11:23 AM
 */

#include "gpu.h"
#include <mpi.h>

gpu::gpu() {

    setDeviceCount();
    if (present){
        if (deviceCount == 0) {
	    gpu(false);
	} else {
	    setDeviceProperties();
	}
    } else {
	    deviceCount = 0;
	    name = new string[1];
	    name[0]=" ";
	    major = new int[1];
	    major[0]=0;
	    minor = new int[1];
	    minor[0]=0;
	    totalGlobalMem = new unsigned int[1];
	    totalGlobalMem[0]=0;
	    multiProcessorCount = new int[1];
	    multiProcessorCount[0]=0;
	    numCores = new int[1];
	    numCores[0]=0;
	    totalConstMem = new unsigned int[1];
	    totalConstMem[0]=0;
	    sharedMemPerBlock = new unsigned int[1];
	    sharedMemPerBlock[0]=0;
	    regsPerBlock = new int[1];
	    regsPerBlock[0]=0;
	    warpSize = new int[1];
	    warpSize[0]=0;
	    maxThreadsPerBlock = new int[1];
	    maxThreadsPerBlock[0]=0;
	    maxThreadsDim0 = new int[1];
	    maxThreadsDim0[0]=0;
	    maxThreadsDim1 = new int[1];
	    maxThreadsDim1[0]=0;
	    maxThreadsDim2 = new int[1];
	    maxThreadsDim2[0]=0;
	    maxGridSize0 = new int[1];
	    maxGridSize0[0]=0;
	    maxGridSize1 = new int[1];
	    maxGridSize1[0]=0;
	    maxGridSize2 = new int[1];
	    maxGridSize2[0]=0;
	    memPitch = new unsigned int[1];
	    memPitch[0]=0;
	    textureAlignment = new unsigned int[1];
	    textureAlignment[0]=0;
	    clockRate = new float[1];
	    clockRate[0]=0;
	    deviceOverlap = new bool[1];
	    deviceOverlap[0]=0;
    }
    
    setNatr();
    setValueatr();
    setNameatr();    
}

gpu::gpu(bool verify) {
    if (!verify){
        present=false;
        deviceCount = 0;
        name = new string[1];
        name[0]=" ";
        major = new int[1];
        major[0]=0;
        minor = new int[1];
        minor[0]=0;
        totalGlobalMem = new unsigned int[1];
        totalGlobalMem[0]=0;
        multiProcessorCount = new int[1];
        multiProcessorCount[0]=0;
        numCores = new int[1];
        numCores[0]=0;
        totalConstMem = new unsigned int[1];
        totalConstMem[0]=0;
        sharedMemPerBlock = new unsigned int[1];
        sharedMemPerBlock[0]=0;
        regsPerBlock = new int[1];
        regsPerBlock[0]=0;
        warpSize = new int[1];
        warpSize[0]=0;
        maxThreadsPerBlock = new int[1];
        maxThreadsPerBlock[0]=0;
        maxThreadsDim0 = new int[1];
        maxThreadsDim0[0]=0;
        maxThreadsDim1 = new int[1];
        maxThreadsDim1[0]=0;
        maxThreadsDim2 = new int[1];
        maxThreadsDim2[0]=0;
        maxGridSize0 = new int[1];
        maxGridSize0[0]=0;
        maxGridSize1 = new int[1];
        maxGridSize1[0]=0;
        maxGridSize2 = new int[1];
        maxGridSize2[0]=0;
        memPitch = new unsigned int[1];
        memPitch[0]=0;
        textureAlignment = new unsigned int[1];
        textureAlignment[0]=0;
        clockRate = new float[1];
        clockRate[0]=0;
        deviceOverlap = new bool[1];
        deviceOverlap[0]=0;
    } else {
	gpu();
    }   
}

gpu::gpu(void *buf, int size){
    int offset=0,aux,nelem=1;
    int auxsize;
    MPI::INT.Unpack(buf,size,&aux,1,offset,MPI::COMM_WORLD);
    if(aux==1) present=true;
    else present=false;
    MPI::INT.Unpack(buf,size,&deviceCount,1,offset,MPI::COMM_WORLD);
    if(deviceCount!=0) nelem=deviceCount;
    name = new string[nelem];
    major = new int[nelem];
    minor = new int[nelem];
    totalGlobalMem = new unsigned int[nelem];
    multiProcessorCount = new int[nelem];
    numCores = new int[nelem];
    totalConstMem = new unsigned int[nelem];
    sharedMemPerBlock = new unsigned int[nelem];
    regsPerBlock = new int[nelem];
    warpSize = new int[nelem];
    maxThreadsPerBlock = new int[nelem];
    maxThreadsDim0 = new int[nelem];
    maxThreadsDim1 = new int[nelem];
    maxThreadsDim2 = new int[nelem];
    maxGridSize0 = new int[nelem];
    maxGridSize1 = new int[nelem];
    maxGridSize2 = new int[nelem];
    memPitch = new unsigned int[nelem];
    textureAlignment = new unsigned int[nelem];
    clockRate = new float[nelem];
    deviceOverlap = new bool[nelem];

    MPI::INT.Unpack(buf,size,&auxsize,1,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,name,auxsize,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,major,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,minor,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,totalGlobalMem,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,multiProcessorCount,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,numCores,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,totalConstMem,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,sharedMemPerBlock,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,regsPerBlock,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,warpSize,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,maxThreadsPerBlock,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,maxThreadsDim0,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,maxThreadsDim1,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,maxThreadsDim2,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,maxGridSize0,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,maxGridSize1,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,maxGridSize2,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,memPitch,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,textureAlignment,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,clockRate,nelem,offset,MPI::COMM_WORLD);
    MPI::INT.Unpack(buf,size,deviceOverlap,nelem,offset,MPI::COMM_WORLD);
}

void gpu::pack(void *buf, int size){
    int offset=0,aux,nelem=1;
    int auxsize=name[0].length();
    if(present) aux=1;
    else aux=0;
    MPI::INT.Pack(&aux,1,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(&deviceCount,1,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(&auxsize,1,buf,size,offset,MPI::COMM_WORLD);
    if(deviceCount!=0) nelem=deviceCount;
    MPI::INT.Pack(name,auxsize*nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(major,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(minor,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(totalGlobalMem,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(multiProcessorCount,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(numCores,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(totalConstMem,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(sharedMemPerBlock,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(regsPerBlock,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(warpSize,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(maxThreadsPerBlock,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(maxThreadsDim0,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(maxThreadsDim1,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(maxThreadsDim2,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(maxGridSize0,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(maxGridSize1,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(maxGridSize2,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(memPitch,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(textureAlignment,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(clockRate,nelem,buf,size,offset,MPI::COMM_WORLD);
    MPI::INT.Pack(deviceOverlap,nelem,buf,size,offset,MPI::COMM_WORLD);
}

void gpu::complete(){
    setNatr();
    setValueatr();
    setNameatr();      
}

void gpu::setPresent(){
    int auxsystem;
    auxsystem=system("which nvcc > nul 2>&1");
    if (auxsystem==0)
        present=true;
    else
        present=false;
}

bool gpu::getPresent(){
    return present;
}

void gpu::setDeviceProperties(){
#if CUDA
    int dev;
    hipDeviceProp_t deviceProp;

    name = new string[deviceCount];
    major = new int[deviceCount];
    minor = new int[deviceCount];
    totalGlobalMem = new unsigned int[deviceCount];
    multiProcessorCount = new int[deviceCount];
    numCores = new int[deviceCount];
    totalConstMem = new unsigned int[deviceCount];
    sharedMemPerBlock = new unsigned int[deviceCount];
    regsPerBlock = new int[deviceCount];
    warpSize = new int[deviceCount];
    maxThreadsPerBlock = new int[deviceCount];
    maxThreadsDim0 = new int[deviceCount];
    maxThreadsDim1 = new int[deviceCount];
    maxThreadsDim2 = new int[deviceCount];
    maxGridSize0 = new int[deviceCount];
    maxGridSize1 = new int[deviceCount];
    maxGridSize2 = new int[deviceCount];
    memPitch = new unsigned int[deviceCount];
    textureAlignment = new unsigned int[deviceCount];
    clockRate = new float[deviceCount];
    deviceOverlap = new bool[deviceCount];

    for (dev = 0; dev < deviceCount; ++dev) {
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		gpu(false);
	    }
        }
	name[dev]=deviceProp.name;
	major[dev]=deviceProp.major;
	minor[dev]=deviceProp.minor;
	totalGlobalMem[dev]=(unsigned int)deviceProp.totalGlobalMem;
    #if CUDART_VERSION >= 2000
	multiProcessorCount[dev]=deviceProp.multiProcessorCount;
	numCores[dev]=8 * deviceProp.multiProcessorCount;
    #else
	multiProcessorCount[dev]=0;
	numCores[dev]=0;
    #endif
	totalConstMem[dev]=(unsigned int)deviceProp.totalConstMem;
	sharedMemPerBlock[dev]=(unsigned int)deviceProp.sharedMemPerBlock;
	regsPerBlock[dev]=deviceProp.regsPerBlock;
	warpSize[dev]=deviceProp.warpSize;
	maxThreadsPerBlock[dev]=deviceProp.maxThreadsPerBlock;
	maxThreadsDim0[dev]=deviceProp.maxThreadsDim[0];
	maxThreadsDim1[dev]=deviceProp.maxThreadsDim[1];
	maxThreadsDim2[dev]=deviceProp.maxThreadsDim[2];
	maxGridSize0[dev]=deviceProp.maxGridSize[0];
	maxGridSize1[dev]=deviceProp.maxGridSize[1];
	maxGridSize2[dev]=deviceProp.maxGridSize[2];
	memPitch[dev]=(unsigned int)deviceProp.memPitch;
	textureAlignment[dev]=(unsigned int)deviceProp.textureAlignment;
	clockRate[dev]=deviceProp.clockRate * 1e-6f;
    #if CUDART_VERSION >= 2000
	deviceOverlap[dev]=deviceProp.deviceOverlap;
    #else
	deviceOverlap[dev]=false;
    #endif
    }
#endif
}

void gpu::gpuCopy(gpu aCopiar){
    int dev;
    present=aCopiar.present;
    if(present){
        deviceCount=aCopiar.deviceCount;
        for (dev = 0; dev < deviceCount; ++dev) {
            name[dev]=aCopiar.name[dev];
            major[dev]=aCopiar.major[dev];
            minor[dev]=aCopiar.minor[dev];
            totalGlobalMem[dev]=aCopiar.totalGlobalMem[dev];
            multiProcessorCount[dev]=aCopiar.multiProcessorCount[dev];
            numCores[dev]=aCopiar.multiProcessorCount[dev];
            totalConstMem[dev]=aCopiar.totalConstMem[dev];
            sharedMemPerBlock[dev]=aCopiar.sharedMemPerBlock[dev];
            regsPerBlock[dev]=aCopiar.regsPerBlock[dev];
            warpSize[dev]=aCopiar.warpSize[dev];
            maxThreadsPerBlock[dev]=aCopiar.maxThreadsPerBlock[dev];
            maxThreadsDim0[dev]=aCopiar.maxThreadsDim0[dev];
            maxThreadsDim1[dev]=aCopiar.maxThreadsDim1[dev];
            maxThreadsDim2[dev]=aCopiar.maxThreadsDim2[dev];
            maxGridSize0[dev]=aCopiar.maxGridSize0[dev];
            maxGridSize1[dev]=aCopiar.maxGridSize1[dev];
            maxGridSize2[dev]=aCopiar.maxGridSize2[dev];
            memPitch[dev]=aCopiar.memPitch[dev];
            textureAlignment[dev]=aCopiar.textureAlignment[dev];
            clockRate[dev]=aCopiar.clockRate[dev];
            deviceOverlap[dev]=aCopiar.deviceOverlap[dev];
        }
    }
    complete();    
}

//Para descubrir la clase
void gpu::setNatr(){
    if (present){
	natr=23;
    } else {
        natr=1;
    }
}

void gpu::setValueatr(){
    valueatr = new string[natr];
    
    
    stringstream auxss;

    if(present){
        auxss << present << ",";  
        valueatr[0]=auxss.str();
        auxss.str(string());

        auxss << deviceCount << ",";  
        valueatr[1]=auxss.str();
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << name[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[2]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << major[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[3]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << minor[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[4]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << totalGlobalMem[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[5]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << multiProcessorCount[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[6]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << numCores[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[7]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << totalConstMem[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[8]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << sharedMemPerBlock[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[9]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << regsPerBlock[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[10]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << warpSize[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[11]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << maxThreadsPerBlock[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[12]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << maxThreadsDim0[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[13]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << maxThreadsDim1[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[14]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << maxThreadsDim2[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[15]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << maxGridSize0[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[16]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << maxGridSize1[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[17]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << maxGridSize2[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[18]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << memPitch[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[19]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << textureAlignment[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[20]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << clockRate[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[21]=auxss.str().append("],");
        auxss.str(string());

        auxss << "[";
        for (int i=0; i<deviceCount;i++){
            auxss << deviceOverlap[i];
            if(i!=deviceCount-1)
                auxss << ",";  
        } 
        valueatr[22]=auxss.str().append("]");
        auxss.str(string());
    } else {
        auxss << present ;  
        valueatr[0]=auxss.str();
        auxss.str(string());
    
    }
}

void gpu::setNameatr(){
    nameatr = new string[natr];

    nameatr[0]="present";
    if(present){
        nameatr[1]="deviceCount";
        nameatr[2]="name";
        nameatr[3]="major";
        nameatr[4]="minor";
        nameatr[5]="totalGlobalMem";
        nameatr[6]="multiProcessorCount";
        nameatr[7]="numCores";
        nameatr[8]="totalConstMem";
        nameatr[9]="sharedMemPerBlock";
        nameatr[10]="regsPerBlock";
        nameatr[11]="warpSize";
        nameatr[12]="maxThreadsPerBlock";
        nameatr[13]="maxThreadsDim0";
        nameatr[14]="maxThreadsDim1";
        nameatr[15]="maxThreadsDim2";
        nameatr[16]="maxGridSize0";
        nameatr[17]="maxGridSize1";
        nameatr[18]="maxGridSize2";
        nameatr[19]="memPitch";
        nameatr[20]="textureAlignment";
        nameatr[21]="clockRate";
        nameatr[22]="deviceOverlap";
    }
}

int gpu::getNatr(){
	return natr;
}

string gpu::getValueatr(int n){
	if (n<getNatr())
		return valueatr[n];
	else
		exit(EXIT_FAILURE);	

	
}
	
string gpu::getNameatr(int n){
	if (n<getNatr())
		return nameatr[n];
	else
		exit(EXIT_FAILURE);	
}

/*gpu::gpu(const gpu& orig) {
}

gpu::~gpu() {
}*/

void gpu::setDeviceCount(){
#if CUDA
   hipGetDeviceCount(&deviceCount);
   if(deviceCount==0) present=false;
   else present=true;
#else
   deviceCount=0;
   present=false;
#endif
}

