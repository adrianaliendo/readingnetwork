/* 
 * File:   gpu.cpp
 * Author: aliendo
 * 
 * Created on 26 de diciembre de 2013, 11:23 AM
 */

#include "gpu.h"

gpu::gpu() {
    setPresent();
    int deviceCount;

    if (present){
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0) {
            cout << "There is no device supporting CUDA" << endl;
	    gpu(false);
	} else {
            cout << "Nro de dispostivos:" << deviceCount << ":" << endl;
	    setDeviceProperties();
	}
    } else {
	    deviceCount = 0;
	    name = new string[1];
	    name[0]=" ";
	    major = new int[1];
	    major[0]=0;
	    minor = new int[1];
	    minor[0]=0;
	    totalGlobalMem = new unsigned int[1];
	    totalGlobalMem[0]=0;
	    multiProcessorCount = new int[1];
	    multiProcessorCount[0]=0;
	    numCores = new int[1];
	    numCores[0]=0;
	    totalConstMem = new unsigned int[1];
	    totalConstMem[0]=0;
	    sharedMemPerBlock = new unsigned int[1];
	    sharedMemPerBlock[0]=0;
	    regsPerBlock = new int[1];
	    regsPerBlock[0]=0;
	    warpSize = new int[1];
	    warpSize[0]=0;
	    maxThreadsPerBlock = new int[1];
	    maxThreadsPerBlock[0]=0;
	    maxThreadsDim0 = new int[1];
	    maxThreadsDim0[0]=0;
	    maxThreadsDim1 = new int[1];
	    maxThreadsDim1[0]=0;
	    maxThreadsDim2 = new int[1];
	    maxThreadsDim2[0]=0;
	    maxGridSize0 = new int[1];
	    maxGridSize0[0]=0;
	    maxGridSize1 = new int[1];
	    maxGridSize1[0]=0;
	    maxGridSize2 = new int[1];
	    maxGridSize2[0]=0;
	    memPitch = new unsigned int[1];
	    memPitch[0]=0;
	    textureAlignment = new unsigned int[1];
	    textureAlignment[0]=0;
	    clockRate = new float[1];
	    clockRate[0]=0;
	    deviceOverlap = new bool[1];
	    deviceOverlap[0]=0;
    }
    
    setNatr();
    setValueatr();
    setNameatr();    
}

gpu::gpu(bool verify) {
    if (!verify){
        present=false;
	    deviceCount = 0;
	    name = new string[1];
	    name[0]=" ";
	    major = new int[1];
	    major[0]=0;
	    minor = new int[1];
	    minor[0]=0;
	    totalGlobalMem = new unsigned int[1];
	    totalGlobalMem[0]=0;
	    multiProcessorCount = new int[1];
	    multiProcessorCount[0]=0;
	    numCores = new int[1];
	    numCores[0]=0;
	    totalConstMem = new unsigned int[1];
	    totalConstMem[0]=0;
	    sharedMemPerBlock = new unsigned int[1];
	    sharedMemPerBlock[0]=0;
	    regsPerBlock = new int[1];
	    regsPerBlock[0]=0;
	    warpSize = new int[1];
	    warpSize[0]=0;
	    maxThreadsPerBlock = new int[1];
	    maxThreadsPerBlock[0]=0;
	    maxThreadsDim0 = new int[1];
	    maxThreadsDim0[0]=0;
	    maxThreadsDim1 = new int[1];
	    maxThreadsDim1[0]=0;
	    maxThreadsDim2 = new int[1];
	    maxThreadsDim2[0]=0;
	    maxGridSize0 = new int[1];
	    maxGridSize0[0]=0;
	    maxGridSize1 = new int[1];
	    maxGridSize1[0]=0;
	    maxGridSize2 = new int[1];
	    maxGridSize2[0]=0;
	    memPitch = new unsigned int[1];
	    memPitch[0]=0;
	    textureAlignment = new unsigned int[1];
	    textureAlignment[0]=0;
	    clockRate = new float[1];
	    clockRate[0]=0;
	    deviceOverlap = new bool[1];
	    deviceOverlap[0]=0;
    }    
}
/*
gpu::gpu(void *buf, int size){
	int offset=0,aux;
	MPI::INT.Unpack(buf,size,&aux,1,offset,MPI::COMM_WORLD);
	if(aux==1) present=true;
	else present=false;
	MPI::INT.Unpack(buf,size,&deviceCount,1,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,name,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,major,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,minor,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,totalGlobalMem,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,multiProcessorCount,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,numCores,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,totalConstMem,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,sharedMemPerBlock,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,regsPerBlock,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,warpSize,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,maxThreadsPerBlock,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,maxThreadsDim0,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,maxThreadsDim1,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,maxThreadsDim2,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,maxGridSize0,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,maxGridSize1,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,maxGridSize2,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,memPitch,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,textureAlignment,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,clockRate,deviceCount,offset,MPI::COMM_WORLD);
	MPI::INT.Unpack(buf,size,deviceOverlap,deviceCount,offset,MPI::COMM_WORLD);
}

void gpu::pack(void *buf, int size){
	int offset=0,aux;
	if(present) aux=1;
	else aux=0;
	MPI::INT.Pack(&aux,1,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(&deviceCount,1,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(name,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(major,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(minor,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(totalGlobalMem,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(multiProcessorCount,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(numCores,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(totalConstMem,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(sharedMemPerBlock,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(regsPerBlock,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(warpSize,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(maxThreadsPerBlock,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(maxThreadsDim0,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(maxThreadsDim1,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(maxThreadsDim2,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(maxGridSize0,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(maxGridSize1,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(maxGridSize2,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(memPitch,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(textureAlignment,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(clockRate,deviceCount,buf,size,offset,MPI::COMM_WORLD);
	MPI::INT.Pack(deviceOverlap,deviceCount,buf,size,offset,MPI::COMM_WORLD);
}
*/
void gpu::complete(){
    setNatr();
    setValueatr();
    setNameatr();      
}

void gpu::setPresent(){
    int auxsystem;
    auxsystem=system("which nvcc > nul 2>&1");
    if (auxsystem==0)
        present=true;
    else
        present=false;
}

bool gpu::getPresent(){
    return present;
}

void gpu::setDeviceProperties(){
    int dev;
    hipDeviceProp_t deviceProp;

    name = new string[deviceCount];
    major = new int[deviceCount];
    minor = new int[deviceCount];
    totalGlobalMem = new unsigned int[deviceCount];
    multiProcessorCount = new int[deviceCount];
    numCores = new int[deviceCount];
    totalConstMem = new unsigned int[deviceCount];
    sharedMemPerBlock = new unsigned int[deviceCount];
    regsPerBlock = new int[deviceCount];
    warpSize = new int[deviceCount];
    maxThreadsPerBlock = new int[deviceCount];
    maxThreadsDim0 = new int[deviceCount];
    maxThreadsDim1 = new int[deviceCount];
    maxThreadsDim2 = new int[deviceCount];
    maxGridSize0 = new int[deviceCount];
    maxGridSize1 = new int[deviceCount];
    maxGridSize2 = new int[deviceCount];
    memPitch = new unsigned int[deviceCount];
    textureAlignment = new unsigned int[deviceCount];
    clockRate = new float[deviceCount];
    deviceOverlap = new bool[deviceCount];

    for (dev = 0; dev < deviceCount; ++dev) {
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999){
                //cout << "There is no device supporting CUDA." << endl;
		gpu(false);
	    }
        }
	name[dev]=deviceProp.name;
	major[dev]=deviceProp.major;
	minor[dev]=deviceProp.minor;
	totalGlobalMem[dev]=(unsigned int)deviceProp.totalGlobalMem;
    #if CUDART_VERSION >= 2000
	multiProcessorCount[dev]=deviceProp.multiProcessorCount;
	numCores[dev]=8 * deviceProp.multiProcessorCount;
    #else
	multiProcessorCount[dev]=0;
	numCores[dev]=0;
    #endif
	totalConstMem[dev]=(unsigned int)deviceProp.totalConstMem;
	sharedMemPerBlock[dev]=(unsigned int)deviceProp.sharedMemPerBlock;
	regsPerBlock[dev]=deviceProp.regsPerBlock;
	warpSize[dev]=deviceProp.warpSize;
	maxThreadsPerBlock[dev]=deviceProp.maxThreadsPerBlock;
	maxThreadsDim0[dev]=deviceProp.maxThreadsDim[0];
	maxThreadsDim1[dev]=deviceProp.maxThreadsDim[1];
	maxThreadsDim2[dev]=deviceProp.maxThreadsDim[2];
	maxGridSize0[dev]=deviceProp.maxGridSize[0];
	maxGridSize1[dev]=deviceProp.maxGridSize[1];
	maxGridSize2[dev]=deviceProp.maxGridSize[2];
	memPitch[dev]=(unsigned int)deviceProp.memPitch;
	textureAlignment[dev]=(unsigned int)deviceProp.textureAlignment;
	clockRate[dev]=deviceProp.clockRate * 1e-6f;
    #if CUDART_VERSION >= 2000
	deviceOverlap[dev]=deviceProp.deviceOverlap;
    #else
	deviceOverlap[dev]=false;
    #endif
    }
}

void gpu::gpuCopy(gpu* aCopiar){
    int dev;
    present=aCopiar->present;
    deviceCount=aCopiar->deviceCount;
    for (dev = 0; dev < deviceCount; ++dev) {
	name[dev]=aCopiar->name[dev];
	major[dev]=aCopiar->major[dev];
	minor[dev]=aCopiar->minor[dev];
	totalGlobalMem[dev]=aCopiar->totalGlobalMem[dev];
	multiProcessorCount[dev]=aCopiar->multiProcessorCount[dev];
	numCores[dev]=aCopiar->multiProcessorCount[dev];
	totalConstMem[dev]=aCopiar->totalConstMem[dev];
	sharedMemPerBlock[dev]=aCopiar->sharedMemPerBlock[dev];
	regsPerBlock[dev]=aCopiar->regsPerBlock[dev];
	warpSize[dev]=aCopiar->warpSize[dev];
	maxThreadsPerBlock[dev]=aCopiar->maxThreadsPerBlock[dev];
	maxThreadsDim0[dev]=aCopiar->maxThreadsDim0[dev];
	maxThreadsDim1[dev]=aCopiar->maxThreadsDim1[dev];
	maxThreadsDim2[dev]=aCopiar->maxThreadsDim2[dev];
	maxGridSize0[dev]=aCopiar->maxGridSize0[dev];
	maxGridSize1[dev]=aCopiar->maxGridSize1[dev];
	maxGridSize2[dev]=aCopiar->maxGridSize2[dev];
	memPitch[dev]=aCopiar->memPitch[dev];
	textureAlignment[dev]=aCopiar->textureAlignment[dev];
	clockRate[dev]=aCopiar->clockRate[dev];
	deviceOverlap[dev]=aCopiar->deviceOverlap[dev];
    }
    complete();    
}

//Para descubrir la clase
void gpu::setNatr(){
	natr=23;
}

void gpu::setValueatr(){
    valueatr = new string[natr];
    
    stringstream auxss;

    auxss << present << ",";  
    valueatr[0]=auxss.str();
    auxss.str(string());

    auxss << deviceCount << ",";  
    valueatr[1]=auxss.str();
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << name[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[2]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << major[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[3]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << minor[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[4]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << totalGlobalMem[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[5]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << multiProcessorCount[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[6]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << numCores[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[7]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << totalConstMem[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[8]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << sharedMemPerBlock[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[9]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << regsPerBlock[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[10]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << warpSize[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[11]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << maxThreadsPerBlock[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[12]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << maxThreadsDim0[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[13]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << maxThreadsDim1[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[14]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << maxThreadsDim2[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[15]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << maxGridSize0[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[16]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << maxGridSize1[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[17]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << maxGridSize2[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[18]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << memPitch[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[19]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << textureAlignment[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[20]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << clockRate[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[21]=auxss.str().append("],");
    auxss.str(string());

    auxss << "[";
    for (int i=0; i<deviceCount;i++){
        auxss << deviceOverlap[i];
        if(i!=deviceCount-1)
            auxss << ",";  
    } 
    valueatr[22]=auxss.str().append("],");
    auxss.str(string());

}

void gpu::setNameatr(){
    nameatr = new string[natr];
//Actualizar todo
    /*nameatr[0]="present";
    nameatr[1]="nx";
    nameatr[2]="ny";*/
}

int gpu::getNatr(){
	return natr;
}

string gpu::getValueatr(int n){
	if (n<getNatr())
		return valueatr[n];
	else
		exit(EXIT_FAILURE);	

	
}
	
string gpu::getNameatr(int n){
	if (n<getNatr())
		return nameatr[n];
	else
		exit(EXIT_FAILURE);	
}

/*gpu::gpu(const gpu& orig) {
}

gpu::~gpu() {
}*/

