/* 
 * File:   gpu.cpp
 * Author: aliendo
 * 
 * Created on 26 de diciembre de 2013, 11:23 AM
 */

#include "gpu.h"
/*
gpu::gpu() {
    setPresent();
    int deviceCount;

    if (present){
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0) {
            cout << "There is no device supporting CUDA" << endl;
            gpu(false);
        } else {
            cout << "Nro de dispostivos:" << deviceCount << ":" << endl;
            setDeviceProperties();
        }
    } else {
            deviceCount = 0;
            name = new string[1];
            name[0]=" ";
            major = new int[1];
            major[0]=0;
            minor = new int[1];
            minor[0]=0;
            totalGlobalMem = new unsigned int[1];
            totalGlobalMem[0]=0;
            multiProcessorCount = new int[1];
            multiProcessorCount[0]=0;
            numCores = new int[1];
            numCores[0]=0;
            totalConstMem = new unsigned int[1];
            totalConstMem[0]=0;
            sharedMemPerBlock = new unsigned int[1];
            sharedMemPerBlock[0]=0;
            regsPerBlock = new int[1];
            regsPerBlock[0]=0;
            warpSize = new int[1];
            warpSize[0]=0;
            maxThreadsPerBlock = new int[1];
            maxThreadsPerBlock[0]=0;
            maxThreadsDim0 = new int[1];
            maxThreadsDim0[0]=0;
            maxThreadsDim1 = new int[1];
            maxThreadsDim1[0]=0;
            maxThreadsDim2 = new int[1];
            maxThreadsDim2[0]=0;
            maxGridSize0 = new int[1];
            maxGridSize0[0]=0;
            maxGridSize1 = new int[1];
            maxGridSize1[0]=0;
            maxGridSize2 = new int[1];
            maxGridSize2[0]=0;
            memPitch = new unsigned int[1];
            memPitch[0]=0;
            textureAlignment = new unsigned int[1];
            textureAlignment[0]=0;
            clockRate = new float[1];
            clockRate[0]=0;
            deviceOverlap = new bool[1];
            deviceOverlap[0]=0;
    }

    setNatr();
    setValueatr();
    setNameatr();
}
*/

void gpu::setDeviceProperties(){
    int dev;
    hipDeviceProp_t deviceProp;

    name = new string[deviceCount];
    major = new int[deviceCount];
    minor = new int[deviceCount];
    totalGlobalMem = new unsigned int[deviceCount];
    multiProcessorCount = new int[deviceCount];
    numCores = new int[deviceCount];
    totalConstMem = new unsigned int[deviceCount];
    sharedMemPerBlock = new unsigned int[deviceCount];
    regsPerBlock = new int[deviceCount];
    warpSize = new int[deviceCount];
    maxThreadsPerBlock = new int[deviceCount];
    maxThreadsDim0 = new int[deviceCount];
    maxThreadsDim1 = new int[deviceCount];
    maxThreadsDim2 = new int[deviceCount];
    maxGridSize0 = new int[deviceCount];
    maxGridSize1 = new int[deviceCount];
    maxGridSize2 = new int[deviceCount];
    memPitch = new unsigned int[deviceCount];
    textureAlignment = new unsigned int[deviceCount];
    clockRate = new float[deviceCount];
    deviceOverlap = new bool[deviceCount];

    for (dev = 0; dev < deviceCount; ++dev) {
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999){
                //cout << "There is no device supporting CUDA." << endl;
		gpu(false);
	    }
        }
	name[dev]=deviceProp.name;
	major[dev]=deviceProp.major;
	minor[dev]=deviceProp.minor;
	totalGlobalMem[dev]=(unsigned int)deviceProp.totalGlobalMem;
    #if CUDART_VERSION >= 2000
	multiProcessorCount[dev]=deviceProp.multiProcessorCount;
	numCores[dev]=8 * deviceProp.multiProcessorCount;
    #else
	multiProcessorCount[dev]=0;
	numCores[dev]=0;
    #endif
	totalConstMem[dev]=(unsigned int)deviceProp.totalConstMem;
	sharedMemPerBlock[dev]=(unsigned int)deviceProp.sharedMemPerBlock;
	regsPerBlock[dev]=deviceProp.regsPerBlock;
	warpSize[dev]=deviceProp.warpSize;
	maxThreadsPerBlock[dev]=deviceProp.maxThreadsPerBlock;
	maxThreadsDim0[dev]=deviceProp.maxThreadsDim[0];
	maxThreadsDim1[dev]=deviceProp.maxThreadsDim[1];
	maxThreadsDim2[dev]=deviceProp.maxThreadsDim[2];
	maxGridSize0[dev]=deviceProp.maxGridSize[0];
	maxGridSize1[dev]=deviceProp.maxGridSize[1];
	maxGridSize2[dev]=deviceProp.maxGridSize[2];
	memPitch[dev]=(unsigned int)deviceProp.memPitch;
	textureAlignment[dev]=(unsigned int)deviceProp.textureAlignment;
	clockRate[dev]=deviceProp.clockRate * 1e-6f;
    #if CUDART_VERSION >= 2000
	deviceOverlap[dev]=deviceProp.deviceOverlap;
    #else
	deviceOverlap[dev]=false;
    #endif
    }
}

void gpu::setDeviceCount(){
    if (present){
        hipGetDeviceCount(&deviceCount);
    } else { 
	deviceCount=0;
    }
}
